#include "hip/hip_runtime.h"
#include "benchmark/multi_exp1/multi_exp_gpu.h"

#include "benchmark/multi_exp1/multiply_add.h"
#include "sxt/curve32/operation/add.h"
#include "sxt/curve32/type/element_p3.h"
#include "sxt/memory/management/managed_array.h"
#include "sxt/memory/resource/device_resource.h"

namespace sxt {
constexpr int num_threads_v = 128;

//--------------------------------------------------------------------------------------------------
// compute_reduction 
//--------------------------------------------------------------------------------------------------
__device__ static void compute_reduction(c32t::element_p3& res_mi,
                                         c32t::element_p3* reductions, int mi,
                                         int first, int last) {
  auto tid = threadIdx.x;
  auto& reduction = reductions[tid];
  reduction = c32t::element_p3::identity();
  for (int i=first; i<last; i+=num_threads_v) {
    multiply_add(reduction, mi, i);
  }

  __syncthreads();
  for (int s=num_threads_v/2; s>0; s>>=1) {
    if (tid < s) {
      c32o::add(reduction, reduction, reductions[tid + s]);
    }
    __syncthreads();
  }
  if (tid == 0) {
    res_mi = reduction;
  }
}


//--------------------------------------------------------------------------------------------------
// multi_exp_kernel 
//--------------------------------------------------------------------------------------------------
__global__ static void multi_exp_kernel(c32t::element_p3* res, int n) {
  __shared__ c32t::element_p3 reductions[num_threads_v];
  auto first = threadIdx.x;
  int mi = blockIdx.x;
  compute_reduction(res[mi], reductions, mi, first, n);
}

//--------------------------------------------------------------------------------------------------
// multi_exp_gpu
//--------------------------------------------------------------------------------------------------
void multi_exp_gpu(c32t::element_p3* res, int m, int n) noexcept {
  memmg::managed_array<c32t::element_p3> device_elements(
      m, memr::get_device_resource());

  multi_exp_kernel<<<m, num_threads_v>>>(device_elements.data(), n);

  hipMemcpy(res, device_elements.data(), device_elements.num_bytes(),
             hipMemcpyDeviceToHost);
}
} // namespace sxt

