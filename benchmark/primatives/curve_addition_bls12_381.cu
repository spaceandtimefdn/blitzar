#include "hip/hip_runtime.h"
/** Proofs GPU - Space and Time's cryptographic proof algorithms on the CPU and GPU.
 *
 * Copyright 2023-present Space and Time Labs, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <iostream>
#include <chrono>

#include "sxt/base/num/divide_up.h"
#include "sxt/base/num/fast_random_number_generator.h"
#include "sxt/curve_g1/operation/add.h"
#include "sxt/curve_g1/random/element_p2.h"
#include "sxt/curve_g1/type/element_p2.h"
#include "sxt/memory/management/managed_array.h"
#include "sxt/memory/resource/device_resource.h"

#define MAX_THREADS_PER_BLOCK 256

namespace sxt {
//--------------------------------------------------------------------------------------------------
// vector_add_impl 
//--------------------------------------------------------------------------------------------------
__global__ void vector_add_impl(const cg1t::element_p2* __restrict__ vec_a,
                                const cg1t::element_p2* __restrict__ vec_b, 
                                cg1t::element_p2* __restrict__ vec_r,
                                unsigned n_elements) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n_elements) {
      cg1o::add(vec_r[tid], vec_a[tid], vec_b[tid]);
    }
}

//--------------------------------------------------------------------------------------------------
// vector_add 
//--------------------------------------------------------------------------------------------------
void vector_add(cg1t::element_p2* vec_b, cg1t::element_p2* __restrict__ vec_a, cg1t::element_p2* vec_result, unsigned n_elements) {
    const unsigned threads_per_block = MAX_THREADS_PER_BLOCK;
    const unsigned num_blocks = basn::divide_up(n_elements, threads_per_block);

    vector_add_impl<<<num_blocks, threads_per_block>>>(vec_a, vec_b, vec_result, n_elements);
}

//--------------------------------------------------------------------------------------------------
// init_random_array_impl 
//--------------------------------------------------------------------------------------------------
__global__ static void init_random_array_impl(cg1t::element_p2* __restrict__ rand, unsigned n_elements) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n_elements)
    {
        basn::fast_random_number_generator rng{static_cast<uint64_t>(tid + 1),
                                               static_cast<uint64_t>(n_elements + 1)};
        cg1rn::generate_random_element(rand[tid], rng);
    }
}

//--------------------------------------------------------------------------------------------------
// init_random_array
//--------------------------------------------------------------------------------------------------
void init_random_array(cg1t::element_p2* rand, unsigned n_elements) {
    const unsigned threads_per_block = MAX_THREADS_PER_BLOCK;
    const unsigned num_blocks = basn::divide_up(n_elements, threads_per_block);

    init_random_array_impl<<<num_blocks, threads_per_block>>>(rand, n_elements);
}

//--------------------------------------------------------------------------------------------------
// add
//--------------------------------------------------------------------------------------------------
void add(size_t vector_size, size_t repetitions) noexcept {
  std::cout << "Blitzar benchmark primative: curve element addition" << std::endl;
  std::cout << "Vector size = " << vector_size << std::endl;
  std::cout << "Repetitions = " << repetitions << std::endl;
  
  // Allocate memory for the input and output vectors
  memmg::managed_array<cg1t::element_p2> a(vector_size, memr::get_device_resource());
  memmg::managed_array<cg1t::element_p2> b(vector_size, memr::get_device_resource());
  memmg::managed_array<cg1t::element_p2> ret(vector_size, memr::get_device_resource());

  // Populate the input vectors with random curve elements
  init_random_array(a.data(), vector_size);
  init_random_array(b.data(), vector_size);

  // Warm-up loop
  std::cout << "Starting warm-up" << std::endl;
  for (int i = 0; i < repetitions; i++) {
    vector_add(a.data(), b.data(), ret.data(), vector_size);
  }

  // Benchmarking loop
  std::cout << "Starting benchmarking" << std::endl;
  auto start_time = std::chrono::steady_clock::now();
  for (int i = 0; i < repetitions; i++) {
    vector_add(a.data(), b.data(), ret.data(), vector_size);
  }
  auto end_time = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

  // Report data
  std::cout << "Elapsed time: " << duration.count() << " microseconds" << std::endl;
  double GMPS = 1.0e-9 * repetitions * vector_size / (1.0e-6 * duration.count()) ;
  std::cout << "Performance: " << GMPS << " Giga curve additions Per Second" << std::endl;

  // Copy the result back to the host. Not necessary for performance measurements.
  // memmg::managed_array<cg1t::element_p2> res(vector_size);
  // hipMemcpy(res.data(), ret.data(), ret.num_bytes(), hipMemcpyDeviceToHost);
}
}
