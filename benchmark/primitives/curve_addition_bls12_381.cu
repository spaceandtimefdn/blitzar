#include "hip/hip_runtime.h"
/** Proofs GPU - Space and Time's cryptographic proof algorithms on the CPU and GPU.
 *
 * Copyright 2023-present Space and Time Labs, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <chrono>
#include <print>

#include "sxt/base/num/divide_up.h"
#include "sxt/base/num/fast_random_number_generator.h"
#include "sxt/curve_g1/operation/add.h"
#include "sxt/curve_g1/random/element_p2.h"
#include "sxt/curve_g1/type/element_p2.h"
#include "sxt/memory/management/managed_array.h"
#include "sxt/memory/resource/device_resource.h"

#define MAX_THREADS_PER_BLOCK 256

namespace sxt {
//--------------------------------------------------------------------------------------------------
// vector_add_impl 
//--------------------------------------------------------------------------------------------------
__global__ void vector_add_impl(const cg1t::element_p2* __restrict__ vec_a,
                                const cg1t::element_p2* __restrict__ vec_b, 
                                cg1t::element_p2* __restrict__ vec_r,
                                unsigned n_elements) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n_elements) {
      cg1o::add(vec_r[tid], vec_a[tid], vec_b[tid]);
    }
}

//--------------------------------------------------------------------------------------------------
// vector_add 
//--------------------------------------------------------------------------------------------------
void vector_add(cg1t::element_p2* vec_b, cg1t::element_p2* __restrict__ vec_a, cg1t::element_p2* vec_result, unsigned n_elements) {
    const unsigned threads_per_block = MAX_THREADS_PER_BLOCK;
    const unsigned num_blocks = basn::divide_up(n_elements, threads_per_block);

    vector_add_impl<<<num_blocks, threads_per_block>>>(vec_a, vec_b, vec_result, n_elements);
}

//--------------------------------------------------------------------------------------------------
// init_random_array_impl 
//--------------------------------------------------------------------------------------------------
__global__ static void init_random_array_impl(cg1t::element_p2* __restrict__ rand, unsigned n_elements) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n_elements)
    {
        basn::fast_random_number_generator rng{static_cast<uint64_t>(tid + 1),
                                               static_cast<uint64_t>(n_elements + 1)};
        cg1rn::generate_random_element(rand[tid], rng);
    }
}

//--------------------------------------------------------------------------------------------------
// init_random_array
//--------------------------------------------------------------------------------------------------
void init_random_array(cg1t::element_p2* rand, unsigned n_elements) {
    const unsigned threads_per_block = MAX_THREADS_PER_BLOCK;
    const unsigned num_blocks = basn::divide_up(n_elements, threads_per_block);

    init_random_array_impl<<<num_blocks, threads_per_block>>>(rand, n_elements);
}

//--------------------------------------------------------------------------------------------------
// add_bls12_381_g1_curve_elements
//--------------------------------------------------------------------------------------------------
void add_bls12_381_g1_curve_elements(unsigned vector_size, unsigned repetitions) noexcept {
  std::print("add_bls12_381_g1_curve_elements");
  
  // Allocate memory for the input and output vectors
  memmg::managed_array<cg1t::element_p2> a(vector_size, memr::get_device_resource());
  memmg::managed_array<cg1t::element_p2> b(vector_size, memr::get_device_resource());
  memmg::managed_array<cg1t::element_p2> ret(vector_size, memr::get_device_resource());

  // Populate the input vectors with random curve elements
  init_random_array(a.data(), vector_size);
  init_random_array(b.data(), vector_size);

  // Warm-up loop
  for (unsigned i = 0; i < repetitions; ++i) {
    vector_add(a.data(), b.data(), ret.data(), vector_size);
  }

  // Benchmarking loop
  auto start_time = std::chrono::steady_clock::now();
  for (unsigned i = 0; i < repetitions; ++i) {
    vector_add(a.data(), b.data(), ret.data(), vector_size);
  }
  auto end_time = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

  // Report data
  std::print("Elapsed time: {} milliseconds\n", duration.count());
  auto GMPS = 1.0e-9 * repetitions * vector_size / (1.0e-3 * duration.count());
  std::print("Performance: {} Giga curve additions Per Second\n", GMPS);
}
}
